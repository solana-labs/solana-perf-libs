
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

#ifndef COMMON_CU
#define COMMON_CU

#define BLOCK_SIZE (4 * 1024)

#define CUDA_CHK(ans) { cuda_assert((ans), __FILE__, __LINE__); }

inline void cuda_assert(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr,"ERR: %s %s %d\n", hipGetErrorString(err), file, line);
        assert(0);
    }
}

#endif
