#include "chacha.h"

#define ROTL32(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define ROTATE(v, c) ROTL32((v), (c))

#define XOR(v, w) ((v) ^ (w))

#define PLUS(x, y) ((x) + (y))

#define U32TO8_LITTLE(p, v) \
{ (p)[0] = ((v)      ) & 0xff; (p)[1] = ((v) >>  8) & 0xff; \
  (p)[2] = ((v) >> 16) & 0xff; (p)[3] = ((v) >> 24) & 0xff; }

#define U8TO32_LITTLE(p)   \
     (((u32)((p)[0])      ) | ((u32)((p)[1]) <<  8) | \
      ((u32)((p)[2]) << 16) | ((u32)((p)[3]) << 24)   )

#define QUARTERROUND(a,b,c,d) \
  x[a] = PLUS(x[a],x[b]); x[d] = ROTATE(XOR(x[d],x[a]),16); \
  x[c] = PLUS(x[c],x[d]); x[b] = ROTATE(XOR(x[b],x[c]),12); \
  x[a] = PLUS(x[a],x[b]); x[d] = ROTATE(XOR(x[d],x[a]), 8); \
  x[c] = PLUS(x[c],x[d]); x[b] = ROTATE(XOR(x[b],x[c]), 7);

#ifdef __CUDA_ARCH__
#define SIGMA_DEF __device__ __constant__
#else
#define SIGMA_DEF
#endif

// sigma contains the ChaCha constants, which happen to be an ASCII string.
static const uint8_t SIGMA_DEF sigma[16] = { 'e', 'x', 'p', 'a', 'n', 'd', ' ', '3',
                                             '2', '-', 'b', 'y', 't', 'e', ' ', 'k' };

static void __host__ __device__ chacha20_encrypt(const u32 input[16],
                                                 unsigned char output[64],
                                                 int num_rounds)
{
    u32 x[16];
    int i;
    memcpy(x, input, sizeof(u32) * 16);
    for (i = num_rounds; i > 0; i -= 2) {
        QUARTERROUND( 0, 4, 8,12)
        QUARTERROUND( 1, 5, 9,13)
        QUARTERROUND( 2, 6,10,14)
        QUARTERROUND( 3, 7,11,15)
        QUARTERROUND( 0, 5,10,15)
        QUARTERROUND( 1, 6,11,12)
        QUARTERROUND( 2, 7, 8,13)
        QUARTERROUND( 3, 4, 9,14)
    }
    for (i = 0; i < 16; ++i) {
        x[i] = PLUS(x[i], input[i]);
    }
    for (i = 0; i < 16; ++i) {
        U32TO8_LITTLE(output + 4 * i, x[i]);
    }
}

void __host__ __device__ chacha20_ctr_encrypt(const uint8_t *in, uint8_t *out, size_t in_len,
                                              const uint8_t key[CHACHA_KEY_SIZE],
                                              const uint8_t nonce[CHACHA_NONCE_SIZE],
                                              uint32_t counter)
{
  uint32_t input[16];
  uint8_t buf[64];
  size_t todo, i;

  input[0] = U8TO32_LITTLE(sigma + 0);
  input[1] = U8TO32_LITTLE(sigma + 4);
  input[2] = U8TO32_LITTLE(sigma + 8);
  input[3] = U8TO32_LITTLE(sigma + 12);

  input[4] = U8TO32_LITTLE(key + 0);
  input[5] = U8TO32_LITTLE(key + 4);
  input[6] = U8TO32_LITTLE(key + 8);
  input[7] = U8TO32_LITTLE(key + 12);

  input[8] = U8TO32_LITTLE(key + 16);
  input[9] = U8TO32_LITTLE(key + 20);
  input[10] = U8TO32_LITTLE(key + 24);
  input[11] = U8TO32_LITTLE(key + 28);

  input[12] = counter;
  input[13] = U8TO32_LITTLE(nonce + 0);
  input[14] = U8TO32_LITTLE(nonce + 4);
  input[15] = U8TO32_LITTLE(nonce + 8);

  while (in_len > 0) {
    todo = sizeof(buf);
    if (in_len < todo) {
      todo = in_len;
    }

    chacha20_encrypt(input, buf, CHACHA_ROUNDS);
    for (i = 0; i < todo; i++) {
      out[i] = in[i] ^ buf[i];
    }

    out += todo;
    in += todo;
    in_len -= todo;

    input[12]++;
  }
}


