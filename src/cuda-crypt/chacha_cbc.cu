#include "hip/hip_runtime.h"
#include "common.cu"
#include "chacha.h"
#include "modes_lcl.h"
#include "perftime.h"
#include <algorithm>
#include "chacha20_core.cu"
#include "gpu_common.h"

#if !defined(STRICT_ALIGNMENT) && !defined(PEDANTIC)
# define STRICT_ALIGNMENT 0
#endif

__host__ __device__ void cuda_chacha20_cbc128_encrypt(const unsigned char* in, unsigned char* out,
                                                      uint32_t len, const uint8_t* key,
                                                      unsigned char* ivec)
{
    size_t n;
    unsigned char *iv = ivec;

    if (len == 0) {
        return;
    }

#if !defined(OPENSSL_SMALL_FOOTPRINT)
    if (STRICT_ALIGNMENT &&
        ((size_t)in | (size_t)out | (size_t)ivec) % sizeof(size_t) != 0) {
        while (len >= CHACHA_BLOCK_SIZE) {
            for (n = 0; n < CHACHA_BLOCK_SIZE; ++n) {
                out[n] = in[n] ^ iv[n];
                //printf("%x ", out[n]);
            }
            chacha20_encrypt((const u32*)out, out, CHACHA_ROUNDS);
            iv = out;
            len -= CHACHA_BLOCK_SIZE;
            in += CHACHA_BLOCK_SIZE;
            out += CHACHA_BLOCK_SIZE;
        }
    } else {
        while (len >= CHACHA_BLOCK_SIZE) {
            for (n = 0; n < CHACHA_BLOCK_SIZE; n += sizeof(size_t)) {
                *(size_t *)(out + n) =
                    *(size_t *)(in + n) ^ *(size_t *)(iv + n);
                //printf("%zu ", *(size_t *)(iv + n));
            }
            chacha20_encrypt((const u32*)out, out, CHACHA_ROUNDS);
            iv = out;
            len -= CHACHA_BLOCK_SIZE;
            in += CHACHA_BLOCK_SIZE;
            out += CHACHA_BLOCK_SIZE;
        }
    }
#endif
    while (len) {
        for (n = 0; n < CHACHA_BLOCK_SIZE && n < len; ++n) {
            out[n] = in[n] ^ iv[n];
        }
        for (; n < CHACHA_BLOCK_SIZE; ++n) {
            out[n] = iv[n];
        }
        chacha20_encrypt((const u32*)out, out, CHACHA_ROUNDS);
        iv = out;
        if (len <= CHACHA_BLOCK_SIZE) {
            break;
        }
        len -= CHACHA_BLOCK_SIZE;
        in += CHACHA_BLOCK_SIZE;
        out += CHACHA_BLOCK_SIZE;
    }
    memcpy(ivec, iv, CHACHA_BLOCK_SIZE);

}

void cuda_chacha20_cbc_encrypt(const uint8_t *in, uint8_t *out, size_t in_len,
                               const uint8_t key[CHACHA_KEY_SIZE], uint8_t* ivec)
{
    cuda_chacha20_cbc128_encrypt(in, out, in_len, key, ivec);
}

__global__ void chacha20_cbc128_encrypt_kernel(const unsigned char* input, unsigned char* output,
                                               size_t length, const uint8_t* keys,
                                               unsigned char* ivec, uint32_t num_keys)
{
    size_t i = (size_t)(blockIdx.x * blockDim.x + threadIdx.x);

    if (i < num_keys) {
        cuda_chacha20_cbc128_encrypt(input, &output[i * length], length, &keys[i], &ivec[i * CHACHA_BLOCK_SIZE]);
    }
}

#include "sha256.cu"

__global__ void init_sha256_state_kernel(hash_state* sha_state, uint32_t num_keys)
{
    size_t i = (size_t)(blockIdx.x * blockDim.x + threadIdx.x);
    if (i < num_keys) {
        sha256_init(&sha_state[i]);
    }
}

__global__ void end_sha256_state_kernel(hash_state* sha_state, uint8_t* out_state, uint32_t num_keys) {
    size_t i = (size_t)(blockIdx.x * blockDim.x + threadIdx.x);
    if (i < num_keys) {
        sha256_done(&sha_state[i], &out_state[i * SHA256_BLOCK_SIZE]);
    }
}

__global__ void chacha20_cbc128_encrypt_sample_kernel(const uint8_t* input,
                                                      uint8_t* output,
                                                      size_t length,
                                                      const uint8_t* keys,
                                                      uint8_t* ivec,
                                                      uint32_t num_keys,
                                                      hash_state* sha_state,
                                                      uint64_t* sample_idx,
                                                      uint32_t sample_len,
                                                      uint64_t block_offset)
{
    size_t i = (size_t)(blockIdx.x * blockDim.x + threadIdx.x);

    if (i < num_keys) {
        uint8_t* t_output = &output[i * BLOCK_SIZE];
        cuda_chacha20_cbc128_encrypt(input, t_output, length, &keys[i * CHACHA_KEY_SIZE], &ivec[i * CHACHA_BLOCK_SIZE]);

        for (uint32_t j = 0; j < sample_len; j++) {
            uint64_t cur_sample = sample_idx[j] * SAMPLE_SIZE;
            if (cur_sample >= block_offset && cur_sample < (block_offset + length)) {
                sha256_process(&sha_state[i], &t_output[cur_sample - block_offset], SAMPLE_SIZE);
            }
        }
    }
}


__global__ void chacha_ctr_encrypt_kernel(const unsigned char* input, unsigned char* output,
                                          size_t length, const uint8_t* keys,
                                          unsigned char* nonces, uint32_t num_keys,
                                          unsigned char* sha_state,
                                          uint32_t* sample_idx,
                                          uint32_t sample_len,
                                          uint32_t block_offset)
{
    size_t i = (size_t)(blockIdx.x * blockDim.x + threadIdx.x);

    if (i < num_keys) {
        chacha20_ctr_encrypt(input, &output[i * length], length, &keys[i * CHACHA_KEY_SIZE], &nonces[i * CHACHA_NONCE_SIZE], 0);
    }
}


void chacha_cbc_encrypt_many(const unsigned char *in, unsigned char *out,
                             size_t length, const uint8_t *keys,
                             uint8_t* ivec,
                             uint32_t num_keys,
                             float* time_us)
{
    if (length < BLOCK_SIZE) {
        printf("ERROR! block size(%d) > length(%zu)\n", BLOCK_SIZE, length);
        return;
    }
    uint8_t* in_device = NULL;
    uint8_t* in_device0 = NULL;
    uint8_t* in_device1 = NULL;
    uint8_t* keys_device = NULL;
    uint8_t* output_device = NULL;
    uint8_t* output_device0 = NULL;
    uint8_t* output_device1 = NULL;
    uint8_t* ivec_device = NULL;

    CUDA_CHK(hipMalloc(&in_device0, BLOCK_SIZE));
    CUDA_CHK(hipMalloc(&in_device1, BLOCK_SIZE));

    size_t keys_size = CHACHA_KEY_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&keys_device, keys_size));
    CUDA_CHK(hipMemcpy(keys_device, keys, keys_size, hipMemcpyHostToDevice));

    size_t ivec_size = CHACHA_BLOCK_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&ivec_device, ivec_size));
    CUDA_CHK(hipMemcpy(ivec_device, ivec, ivec_size, hipMemcpyHostToDevice));

    size_t output_size = (size_t)num_keys * (size_t)BLOCK_SIZE;
    CUDA_CHK(hipMalloc(&output_device0, output_size));
    CUDA_CHK(hipMalloc(&output_device1, output_size));

    int num_threads_per_block = 64;
    int num_blocks = ROUND_UP_DIV(num_keys, num_threads_per_block);

    perftime_t start, end;

    get_time(&start);

    hipStream_t stream, stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    ssize_t slength = length;
    size_t num_data_blocks = (length + BLOCK_SIZE - 1) / (BLOCK_SIZE);

    LOG("num_blocks: %d threads_per_block: %d keys size: %zu in: %p ind0: %p ind1: %p output_size: %zu num_data_blocks: %zu\n",
                    num_blocks, num_threads_per_block, keys_size, in, in_device0, in_device1, output_size, num_data_blocks);

    for (uint32_t i = 0;; i++) {
        //if (i & 0x1) {
        if (0) {
            in_device = in_device1;
            output_device = output_device1;
            stream = stream1;
        } else {
            in_device = in_device0;
            output_device = output_device0;
            stream = stream0;
        }
        size_t size = std::min(slength, (ssize_t)BLOCK_SIZE);
        //printf("copying to in_device: %p in: %p size: %zu num_data_blocks: %zu\n", in_device, in, size, num_data_blocks);
        CUDA_CHK(hipMemcpyAsync(in_device, in, size, hipMemcpyHostToDevice, stream));

        chacha20_cbc128_encrypt_kernel<<<num_blocks, num_threads_per_block, 0, stream>>>(
                            in_device, output_device, size,
                            keys_device, ivec_device, num_keys);
//#define DO_COPY
#ifdef DO_COPY
        for (uint32_t j = 0; j < num_keys; j++) {
            size_t block_offset = j * length + i * BLOCK_SIZE;
            size_t out_offset = j * size;
            //printf("i: %d j: %d copy %zi b block offset: %zu output offset: %zu num_data_blocks: %zu\n",
            //                i, j, size, block_offset, out_offset, num_data_blocks);
            CUDA_CHK(hipMemcpy(&out[block_offset], &output_device[out_offset], size, hipMemcpyDeviceToHost));
        }
#endif

        slength -= BLOCK_SIZE;
        in += BLOCK_SIZE;
        if (slength <= 0) {
            break;
        }
    }

    CUDA_CHK(hipMemcpy(ivec, ivec_device, ivec_size, hipMemcpyDeviceToHost));

    get_time(&end);
    *time_us = get_diff(&start, &end);

    //printf("gpu time: %f us\n", get_diff(&start, &end));
}

void chacha_init_sha_state(void* sha_state_arg, uint32_t num_keys)
{
    LOG("sizeof(hash_state) %zu\n", sizeof(hash_state));
    hash_state* sha_state = (hash_state*)sha_state_arg;
    hash_state* sha_state_device = NULL;
    size_t sha_state_size = num_keys * sizeof(hash_state);
    CUDA_CHK(hipMalloc(&sha_state_device, sha_state_size));

    int num_threads_per_block = 64;
    int num_blocks = ROUND_UP_DIV(num_keys, num_threads_per_block);
    init_sha256_state_kernel<<<num_blocks, num_threads_per_block>>>(sha_state_device, num_keys);

    CUDA_CHK(hipMemcpy(sha_state, sha_state_device, sha_state_size, hipMemcpyDeviceToHost));

    hipFree(sha_state_device);
}

void chacha_end_sha_state(const void* sha_state_arg, uint8_t* out, uint32_t num_keys)
{
    const hash_state* sha_state = (const hash_state*)sha_state_arg;
    uint8_t* out_device = NULL;
    hash_state* sha_state_device = NULL;
    size_t sha_state_size = num_keys * sizeof(hash_state);
    CUDA_CHK(hipMalloc(&sha_state_device, sha_state_size));

    size_t out_size = SHA256_BLOCK_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&out_device, out_size));

    CUDA_CHK(hipMemcpy(sha_state_device, sha_state, sha_state_size, hipMemcpyHostToDevice));

    int num_threads_per_block = 64;
    int num_blocks = ROUND_UP_DIV(num_keys, num_threads_per_block);
    end_sha256_state_kernel<<<num_blocks, num_threads_per_block>>>(sha_state_device, out_device, num_keys);

    CUDA_CHK(hipMemcpy(out, out_device, out_size, hipMemcpyDeviceToHost));

    hipFree(sha_state_device);
    hipFree(out_device);
}

void chacha_cbc_encrypt_many_sample(const uint8_t* in,
                                    void* sha_state_arg,
                                    size_t length,
                                    const uint8_t* keys,
                                    uint8_t* ivecs,
                                    uint32_t num_keys,
                                    const uint64_t* samples,
                                    uint32_t num_samples,
                                    uint64_t starting_block_offset,
                                    float* time_us)
{
    hash_state* sha_state = (hash_state*)sha_state_arg;
    LOG("encrypt_many_sample in: %p len: %zu\n", in, length);
    LOG("    ivecs: %p num_keys: %d\n", ivecs, num_keys);
    uint8_t* in_device = NULL;
    uint8_t* in_device0 = NULL;
    uint8_t* in_device1 = NULL;
    uint8_t* output_device = NULL;
    uint8_t* output_device0 = NULL;
    uint8_t* output_device1 = NULL;
    uint8_t* keys_device = NULL;
    uint8_t* ivec_device = NULL;

    hash_state* sha_state_device = NULL;

    uint64_t* samples_device = NULL;

    LOG("samples:");
    for (uint32_t i = 0; i < num_samples; i++) {
        LOG("%ld ", samples[i]);
    }
    LOG("\n");

    size_t samples_size = sizeof(uint64_t) * num_samples;
    CUDA_CHK(hipMalloc(&samples_device, samples_size));
    CUDA_CHK(hipMemcpy(samples_device, samples, samples_size, hipMemcpyHostToDevice));

    CUDA_CHK(hipMalloc(&in_device0, BLOCK_SIZE));
    CUDA_CHK(hipMalloc(&in_device1, BLOCK_SIZE));

    size_t keys_size = CHACHA_KEY_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&keys_device, keys_size));
    CUDA_CHK(hipMemcpy(keys_device, keys, keys_size, hipMemcpyHostToDevice));

    size_t ivec_size = CHACHA_BLOCK_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&ivec_device, ivec_size));
    CUDA_CHK(hipMemcpy(ivec_device, ivecs, ivec_size, hipMemcpyHostToDevice));

    size_t output_size = (size_t)num_keys * (size_t)BLOCK_SIZE;
    CUDA_CHK(hipMalloc(&output_device0, output_size));
    CUDA_CHK(hipMalloc(&output_device1, output_size));

    size_t sha_state_size = num_keys * sizeof(hash_state);
    CUDA_CHK(hipMalloc(&sha_state_device, sha_state_size));

    CUDA_CHK(hipMemcpy(sha_state_device, sha_state, sha_state_size, hipMemcpyHostToDevice));

    int num_threads_per_block = 64;
    int num_blocks = ROUND_UP_DIV(num_keys, num_threads_per_block);

    perftime_t start, end;

    get_time(&start);

    //hipStream_t stream, stream0, stream1;
    //hipStreamCreate(&stream0);
    //hipStreamCreate(&stream1);

    ssize_t slength = length;
    size_t num_data_blocks = std::max(1ul, (length + BLOCK_SIZE - 1) / (BLOCK_SIZE));

    LOG("num_blocks: %d threads_per_block: %d keys size: %zu in: %p ind0: %p ind1: %p output_size: %zu num_data_blocks: %zu\n",
                    num_blocks, num_threads_per_block, keys_size, in, in_device0, in_device1, output_size, num_data_blocks);

    LOG("ivecs:\n");
    for (size_t nkey = 0; nkey < num_keys; nkey++) {
        LOG("ivec: %zu:\n", nkey);
        for (size_t i = 0; i < CHACHA_BLOCK_SIZE; i++) {
            LOG("%d ", ivecs[nkey * CHACHA_BLOCK_SIZE + i]);
        }
        LOG("\n");
    }
    LOG("\n");

    for (uint32_t i = 0;; i++) {
        //if (i & 0x1) {
        if (0) {
            in_device = in_device1;
            output_device = output_device1;
            //stream = stream1;
        } else {
            in_device = in_device0;
            output_device = output_device0;
            //stream = stream0;
        }
        size_t size = std::min(slength, (ssize_t)BLOCK_SIZE);
        LOG("copying to in_device: %p in: %p size: %zu num_data_blocks: %zu\n", in_device, in, size, num_data_blocks);
        CUDA_CHK(hipMemcpy(in_device, in, size, hipMemcpyHostToDevice));

        LOG("done copying to in_device\n");
        chacha20_cbc128_encrypt_sample_kernel<<<num_blocks, num_threads_per_block>>>(
                            in_device, output_device, size,
                            keys_device, ivec_device, num_keys,
                            sha_state_device,
                            samples_device,
                            num_samples,
                            i * BLOCK_SIZE + starting_block_offset);
//#define DO_COPY
#ifdef DO_COPY
        LOG("doing copy... i=%d\n", i);
        for (uint32_t j = 0; j < num_keys; j++) {
            size_t block_offset = j * length + i * BLOCK_SIZE;
            size_t out_offset = j * size;
            LOG("i: %d j: %d copy %zi b block offset: %zu output offset: %zu num_data_blocks: %zu\n",
                            i, j, size, block_offset, out_offset, num_data_blocks);
            CUDA_CHK(hipMemcpy(&out[block_offset], &output_device[out_offset], size, hipMemcpyDeviceToHost));
        }
#endif

        slength -= BLOCK_SIZE;
        in += BLOCK_SIZE;
        if (slength <= 0) {
            break;
        }
    }

    CUDA_CHK(hipMemcpy(ivecs, ivec_device, ivec_size, hipMemcpyDeviceToHost));
    CUDA_CHK(hipMemcpy(sha_state, sha_state_device, sha_state_size, hipMemcpyDeviceToHost));

    get_time(&end);
    *time_us = get_diff(&start, &end);

    //LOG("gpu time: %f us\n", get_diff(&start, &end));

    CUDA_CHK(hipFree(samples_device));
    CUDA_CHK(hipFree(in_device0));
    CUDA_CHK(hipFree(in_device1));
    CUDA_CHK(hipFree(keys_device));
    CUDA_CHK(hipFree(ivec_device));
    CUDA_CHK(hipFree(output_device0));
    CUDA_CHK(hipFree(output_device1));
    CUDA_CHK(hipFree(sha_state_device));
}



void chacha_ctr_encrypt_many(const unsigned char *in, unsigned char *out,
                             size_t length,
                             const uint8_t *keys,
                             const uint8_t* nonces,
                             uint32_t num_keys,
                             float* time_us)
{
    if (length < BLOCK_SIZE) {
        printf("ERROR! block size(%d) > length(%zu)\n", BLOCK_SIZE, length);
        return;
    }
    uint8_t* in_device = NULL;
    uint8_t* in_device0 = NULL;
    uint8_t* in_device1 = NULL;
    uint8_t* keys_device = NULL;
    uint8_t* output_device = NULL;
    uint8_t* output_device0 = NULL;
    uint8_t* output_device1 = NULL;
    uint8_t* nonces_device = NULL;

    uint8_t* sha_state_device = NULL;

    uint32_t sample_len = 0;
    uint32_t* samples_device = NULL;

    CUDA_CHK(hipMalloc(&in_device0, BLOCK_SIZE));
    CUDA_CHK(hipMalloc(&in_device1, BLOCK_SIZE));

    size_t keys_size = CHACHA_KEY_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&keys_device, keys_size));
    CUDA_CHK(hipMemcpy(keys_device, keys, keys_size, hipMemcpyHostToDevice));

    size_t nonces_size = CHACHA_NONCE_SIZE * num_keys;
    CUDA_CHK(hipMalloc(&nonces_device, nonces_size));
    CUDA_CHK(hipMemcpy(nonces_device, nonces, nonces_size, hipMemcpyHostToDevice));

    size_t output_size = (size_t)num_keys * (size_t)BLOCK_SIZE;
    CUDA_CHK(hipMalloc(&output_device0, output_size));
    CUDA_CHK(hipMalloc(&output_device1, output_size));

    int num_threads_per_block = 64;
    int num_blocks = ROUND_UP_DIV(num_keys, num_threads_per_block);

    perftime_t start, end;

    get_time(&start);

    hipStream_t stream, stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    ssize_t slength = length;
    size_t num_data_blocks = (length + BLOCK_SIZE - 1) / (BLOCK_SIZE);

    LOG("num_blocks: %d threads_per_block: %d keys size: %zu in: %p ind0: %p ind1: %p output_size: %zu num_data_blocks: %zu\n",
                    num_blocks, num_threads_per_block, keys_size, in, in_device0, in_device1, output_size, num_data_blocks);

    for (uint32_t i = 0;; i++) {
        //if (i & 0x1) {
        if (0) {
            in_device = in_device1;
            output_device = output_device1;
            stream = stream1;
        } else {
            in_device = in_device0;
            output_device = output_device0;
            stream = stream0;
        }
        size_t size = std::min(slength, (ssize_t)BLOCK_SIZE);
        //printf("copying to in_device: %p in: %p size: %zu num_data_blocks: %zu\n", in_device, in, size, num_data_blocks);
        CUDA_CHK(hipMemcpyAsync(in_device, in, size, hipMemcpyHostToDevice, stream));

        chacha_ctr_encrypt_kernel<<<num_blocks, num_threads_per_block, 0, stream>>>(
                            in_device, output_device, size,
                            keys_device, nonces_device, num_keys,
                            sha_state_device,
                            samples_device,
                            sample_len,
                            i * BLOCK_SIZE);
#ifdef DO_COPY
        for (uint32_t j = 0; j < num_keys; j++) {
            size_t block_offset = j * length + i * BLOCK_SIZE;
            size_t out_offset = j * size;
            //printf("i: %d j: %d copy %zi b block offset: %zu output offset: %zu num_data_blocks: %zu\n",
            //                i, j, size, block_offset, out_offset, num_data_blocks);
            CUDA_CHK(hipMemcpy(&out[block_offset], &output_device[out_offset], size, hipMemcpyDeviceToHost));
        }
#endif

        slength -= BLOCK_SIZE;
        in += BLOCK_SIZE;
        if (slength <= 0) {
            break;
        }
    }

#ifndef DO_COPY
    CUDA_CHK(hipDeviceSynchronize());
#endif

    get_time(&end);
    *time_us = get_diff(&start, &end);

    //printf("gpu time: %f us\n", get_diff(&start, &end));
}

