#include "hip/hip_runtime.h"
#include "sha512.h"
#include <algorithm>
#include <stdio.h>
#include "ge.cu"
#include "sc.cu"
#include "fe.cu"
#include "seed.cu"
#include "keypair.cu"
#include "sign.cu"
#include "sha512.cu"

#include "ed25519.h"
#include <pthread.h>

#define USE_CLOCK_GETTIME
#include "perftime.h"

static int __host__ __device__ consttime_equal(const unsigned char *x, const unsigned char *y) {
    unsigned char r = 0;

    r = x[0] ^ y[0];
    #define F(i) r |= x[i] ^ y[i]
    F(1);
    F(2);
    F(3);
    F(4);
    F(5);
    F(6);
    F(7);
    F(8);
    F(9);
    F(10);
    F(11);
    F(12);
    F(13);
    F(14);
    F(15);
    F(16);
    F(17);
    F(18);
    F(19);
    F(20);
    F(21);
    F(22);
    F(23);
    F(24);
    F(25);
    F(26);
    F(27);
    F(28);
    F(29);
    F(30);
    F(31);
    #undef F

    return !r;
}

static int __device__ __host__
ed25519_verify_device(const unsigned char *signature,
                      const unsigned char *message,
                      uint32_t message_len,
                      const unsigned char *public_key) {
    unsigned char h[64];
    unsigned char checker[32];
    sha512_context hash;
    ge_p3 A;
    ge_p2 R;

    if (signature[63] & 224) {
        return 0;
    }

    if (ge_frombytes_negate_vartime(&A, public_key) != 0) {
        return 0;
    }

    sha512_init(&hash);
    sha512_update(&hash, signature, 32);
    sha512_update(&hash, public_key, 32);
    sha512_update(&hash, message, message_len);
    sha512_final(&hash, h);
    
    sc_reduce(h);
    ge_double_scalarmult_vartime(&R, h, &A, signature + 32);
    ge_tobytes(checker, &R);

    if (!consttime_equal(checker, signature)) {
        return 0;
    }

    return 1;
}

int 
ed25519_verify(const unsigned char *signature,
               const unsigned char *message,
               uint32_t message_len,
               const unsigned char *public_key) {
    return ed25519_verify_device(signature, message, message_len, public_key);
}

__global__ void ed25519_verify_kernel(const streamer_Packet* packets,
                                      uint32_t public_key_offset,
                                      uint32_t signature_offset,
                                      uint32_t message_start_offset,
                                      uint32_t message_len_offset,
                                      size_t num_keys,
                                      uint8_t* out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_keys) {
        const streamer_Packet* packet = &packets[i];
        uint32_t message_len = ((uint32_t*)&packet->data[message_len_offset])[0];
        if (message_len > message_start_offset) {
            out[i] = ed25519_verify_device(&packet->data[signature_offset],
                                           &packet->data[message_start_offset],
                                           message_len - message_start_offset,
                                           &packet->data[public_key_offset]);
        }
    }
}

typedef struct {
    streamer_Packet* packets;
    uint8_t* out;

    size_t num;
    uint32_t total_packets_len;
    pthread_mutex_t mutex;
} gpu_ctx;

static pthread_mutex_t g_ctx_mutex = PTHREAD_MUTEX_INITIALIZER;

#define MAX_NUM_GPUS 8
#define MAX_QUEUE_SIZE 8

static gpu_ctx g_gpu_ctx[MAX_NUM_GPUS][MAX_QUEUE_SIZE] = {0};
static uint32_t g_cur_gpu = 0;
static uint32_t g_cur_queue[MAX_NUM_GPUS] = {0};
static int32_t g_total_gpus = -1;
static bool g_verbose = false;
#define LOG(...) if (g_verbose) { printf(__VA_ARGS__); }

void ed25519_set_verbose(bool val) {
    g_verbose = val;
}

static bool ed25519_init_locked() {
    if (g_total_gpus == -1) {
        hipGetDeviceCount(&g_total_gpus);
        g_total_gpus = min(8, g_total_gpus);
        LOG("total_gpus: %d\n", g_total_gpus);
        for (int gpu = 0; gpu < g_total_gpus; gpu++) {
            for (int queue = 0; queue < MAX_QUEUE_SIZE; queue++) {
                int err = pthread_mutex_init(&g_gpu_ctx[gpu][queue].mutex, NULL);
                if (err != 0) {
                    fprintf(stderr, "pthread_mutex_init error %d gpu: %d queue: %d\n",
                            err, gpu, queue);
                    g_total_gpus = 0;
                    return false;
                }
            }
        }
    }
    return g_total_gpus > 0;
}

bool ed25519_init() {
    hipFree(0);
    pthread_mutex_lock(&g_ctx_mutex);
    bool success = ed25519_init_locked();
    pthread_mutex_unlock(&g_ctx_mutex);
    return success;
}

void ed25519_verify_many(const gpu_Elems* elems,
                         uint32_t num,
                         uint32_t message_size,
                         uint32_t public_key_offset,
                         uint32_t signature_offset,
                         uint32_t message_start_offset,
                         uint32_t message_len_offset,
                         uint8_t* out)
{
    size_t out_size = 0;
    LOG("Starting verify_many keys: %d\n message_size: %d message_start_offset: %d\n",
	(int)num, message_size, message_start_offset);

    uint32_t total_packets_len = 0;
    uint32_t total_packets = 0;

    if (message_size != sizeof(streamer_Packet)) {
        fprintf(stderr, "ERROR: cuda packet size (%d) doesn't match passed packet size: (%zu)\n",
                        message_size, sizeof(streamer_Packet));
        assert(0);
    }

    for (size_t i = 0; i < num; i++) {
        total_packets += elems[i].num;
        total_packets_len += elems[i].num * sizeof(streamer_Packet);
        out_size += elems[i].num * sizeof(uint8_t);
    }

    if (0 == total_packets) {
        return;
    }

    int32_t cur_gpu, cur_queue;

    LOG("device allocate. packets: %d out: %d\n", total_packets_len, (int)out_size);
    // Device allocate

    pthread_mutex_lock(&g_ctx_mutex);
    if (!ed25519_init_locked()) {
        pthread_mutex_unlock(&g_ctx_mutex);
        LOG("No GPUs, exiting...\n");
        return;
    }
    cur_gpu = g_cur_gpu;
    g_cur_gpu++;
    g_cur_gpu %= g_total_gpus;
    cur_queue = g_cur_queue[cur_gpu];
    g_cur_queue[cur_gpu]++;
    g_cur_queue[cur_gpu] %= MAX_QUEUE_SIZE;
    pthread_mutex_unlock(&g_ctx_mutex);

    gpu_ctx* cur_ctx = &g_gpu_ctx[cur_gpu][cur_queue];
    pthread_mutex_lock(&cur_ctx->mutex);

    hipSetDevice(cur_gpu);

    LOG("cur gpu: %d cur queue: %d\n", cur_gpu, cur_queue);

    if (cur_ctx->packets == NULL ||
        total_packets_len > cur_ctx->total_packets_len) {
        CUDA_CHK(hipFree(cur_ctx->packets));
        CUDA_CHK(hipMalloc(&cur_ctx->packets, total_packets_len));

        cur_ctx->total_packets_len = total_packets_len;
    }

    if (cur_ctx->out == NULL || cur_ctx->num < total_packets) {
        CUDA_CHK(hipFree(cur_ctx->out));
        CUDA_CHK(hipMalloc(&cur_ctx->out, out_size));

        cur_ctx->num = total_packets;
    }

    size_t cur = 0;
    for (size_t i = 0; i < num; i++) {
        LOG("i: %zu size: %zu\n", i, elems[i].num * sizeof(streamer_Packet));
        CUDA_CHK(hipMemcpy(&cur_ctx->packets[cur], elems[i].elems, elems[i].num * sizeof(streamer_Packet), hipMemcpyHostToDevice));
        cur += elems[i].num;

        if (elems[i].num > 0) {
            uint32_t message_len = ((uint32_t*)&elems[i].elems[0].data[message_len_offset])[0];
            LOG("message_len: %d\n", message_len);

            for (size_t j = 0; j < message_len; j++) {
                LOG("%d ", elems[i].elems[0].data[message_start_offset + j]);
            }
        }
    }

    int num_threads_per_block = 64;
    int num_blocks = (total_packets + num_threads_per_block - 1) / num_threads_per_block;
    LOG("num_blocks: %d threads_per_block: %d keys: %d out: %p\n",
           num_blocks, num_threads_per_block, (int)total_packets, out);

    if (num > 0 && elems[0].num > 0) {
        LOG("signature: ");
        for (int i = 0; i < SIG_SIZE; i++) {
            LOG("%d ", elems[0].elems[0].data[signature_offset + i]);
        }
        LOG("\n");

        LOG("pub_key: ");
        for (int i = 0; i < PUB_KEY_SIZE; i++) {
            LOG("%d ", elems[0].elems[0].data[public_key_offset + i]);
        }
        LOG("\n");
    }

    perftime_t start, end;
    get_time(&start);
    ed25519_verify_kernel<<<num_blocks, num_threads_per_block>>>
                            (cur_ctx->packets,
                             public_key_offset,
                             signature_offset,
                             message_start_offset,
                             message_len_offset,
                             cur_ctx->num,
                             cur_ctx->out);
    CUDA_CHK(hipPeekAtLastError());

    hipError_t err = hipMemcpy(out, cur_ctx->out, out_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)  {
        fprintf(stderr, "hipMemcpy(out) error: out = %p cur_ctx->out = %p size = %zu num: %d elems = %p\n",
                        out, cur_ctx->out, out_size, num, elems);
    }
    CUDA_CHK(err);

    pthread_mutex_unlock(&cur_ctx->mutex);

    get_time(&end);
    LOG("time diff: %f\n", get_diff(&start, &end));
}

void ed25519_free_gpu_mem() {
    for (size_t gpu = 0; gpu < MAX_NUM_GPUS; gpu++) {
        for (size_t queue = 0; queue < MAX_QUEUE_SIZE; queue++) {
            gpu_ctx* cur_ctx = &g_gpu_ctx[gpu][queue];
            CUDA_CHK(hipFree(cur_ctx->packets));
            CUDA_CHK(hipFree(cur_ctx->out));
        }
    }
}

// Ensure copyright and license notice is embedded in the binary
const char* ed25519_license() {
   return "Copyright (c) 2018 Solana Labs, Inc. "
	"License AGPLv3: GNU Affero General Public License "
	"<https://www.gnu.org/licenses/agpl-3.0.html>";
}
